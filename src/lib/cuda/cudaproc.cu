#include <hip/hip_runtime.h>

__device__ unsigned char rgb_bound(int v)
{
    return v>255?255:(v<0?0:v);
}

template<typename T>
__device__ T max(T v1, T v2)
{
    return v1>v2?v1:v2;
}

template<typename T>
__device__ T min(T v1, T v2)
{
    return v1<v2?v1:v2;
}

__global__ void yuyv2yuv_kernal(unsigned char *in, unsigned char *out, int w, int h)
{
    int x=blockIdx.x;
    int y=threadIdx.x;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;

    out[tmp*3+dst_offset+0] = in[tmp*2+src_offset+0];
    out[tmp*3+dst_offset+1] = in[tmp*2+src_offset+(int)powf(-1, x&1)];
    out[tmp*3+dst_offset+2] = in[tmp*2+src_offset+2+(int)powf(-1, x&1)];
}

__global__ void yuyv2bgr_kernal(unsigned char *in, unsigned char *out, int w, int h)
{
    int x=blockIdx.x;
    int y=threadIdx.x;
    int tmp = y*w;
    int src_offset = x*2;
    int dst_offset = x*3;

    unsigned char Y = in[tmp*2+src_offset+0];
    unsigned char U = in[tmp*2+src_offset+(int)powf(-1, x&1)];
    unsigned char V = in[tmp*2+src_offset+2+(int)powf(-1, x&1)];
    float r,g,b;

    r = (1.164 * (Y - 16)) + (2.018 * (V - 128));
    g = (1.164 * (Y - 16)) - (0.813 * (U - 128)) - (0.391 * (V - 128));
    b = (1.164 * (Y - 16)) + (1.596 * (U - 128));

    out[tmp*3+dst_offset+2] = rgb_bound(r);
    out[tmp*3+dst_offset+1] = rgb_bound(g);
    out[tmp*3+dst_offset+0] = rgb_bound(b);
}

__global__ void bgr2rgbfp(unsigned char *in, float *rgbfp, int w, int h)
{
    int x=blockIdx.x;
    int y=threadIdx.x;
    int offset = y*w*3+x*3;
    float rf, gf, bf;
    rf = in[offset+2]/255.0f;
    gf = in[offset+1]/255.0f;
    bf = in[offset+0]/255.0f;
    int planesize = w*h;
    int tmp = y*w+x;
    rgbfp[tmp] = rf;
    rgbfp[planesize+tmp] = gf;
    rgbfp[planesize*2+tmp] = bf;
}

__global__ void baygr2bgr_kernal(unsigned char *bayergr, unsigned char *bgr, int w, int h,
    float ds, float rgain, float ggain, float bgain)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int outy = h-y;
    int outx = w-x;
    float r,g,b;
    float hue, sat, val;
    float rn, gn, bn;

    b = bayergr[(y+((y+1)&1))*w+x-(x&1)]*bgain;
    g = bayergr[y*w+x-(x&1)+(y&1)]*ggain;
    r = bayergr[(y-(y&1))*w+x+((x+1)&1)]*rgain;

    float rgbMax = max(max(r,g), b);
    float rgbMin = min(min(r,g), b);
    float delta = rgbMax-rgbMin;

    val = rgbMax;
    if(rgbMax == 0) sat = 0;
    else sat = delta/rgbMax;

    if(delta == 0) hue = 0;
    else
    {
        if(rgbMax == r)
        {
            if(g>=b) hue = 60*(g-b)/delta;
            else hue = 60*(g-b)/delta+360;
        }
        else if(rgbMax == g)
        {
            hue = 60*(b-r)/delta+120;
        }
        else
        {
            hue = 60*(r-g)/delta+240;
        }
    }
    if(ds>=0) sat = sat+(1-sat)*ds;
    else sat = sat+sat*ds;

    int hi = hue/60.0;
    float f = hue/60.0-hi;
    float p = val*(1-sat);
    float q = val*(1-f*sat);
    float t = val*(1-(1-f)*sat);
    switch(hi)
    {
        case 0:
            rn = val; gn = t; bn = p; break;
        case 1:
            rn = q; gn = val; bn = p; break;
        case 2:
            rn = p; gn = val; bn = t; break;
        case 3:
            rn = p; gn = q; bn = val; break;
        case 4:
            rn = t; gn = p; bn = val; break;
        case 5:
            rn = val; gn = p; bn = q; break;
        default:
            break;
    }
    bgr[outy*w*3+outx*3+0] = rgb_bound(bn);
    bgr[outy*w*3+outx*3+1] = rgb_bound(gn);
    bgr[outy*w*3+outx*3+2] = rgb_bound(rn);
}

template<typename T>
__global__ void resize_packed_kernal(T *in, int iw, int ih, T *out, int ow, int oh)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    int offset_out = y*ow*3+x*3;
    float h_scale_rate = (float)ih/oh;
    float w_scale_rate = (float)iw/ow;
    float y_scale = h_scale_rate * y;
    float x_scale = w_scale_rate * x;
    int j = y_scale, i = x_scale;
    float u = y_scale-j, v = x_scale-i;
    int offset_in1 = j*iw*3;
    int offset_in2 = (j+1)*iw*3;
    if(j+1>=ih || i+1>=iw)
    {
        out[offset_out+0] = in[offset_in1+i*3];
        out[offset_out+1] = in[offset_in1+i*3+1];
        out[offset_out+2] = in[offset_in1+i*3+2];
    }
    else
    {
        unsigned char x1,x2,x3,x4;
        x1 = in[offset_in1+i*3];
        x2 = in[offset_in1+(i+1)*3];
        x3 = in[offset_in2+i*3];
        x4 = in[offset_in2+(i+1)*3];
        out[offset_out+0] = ((1-u)*(1-v)*x1+(1-u)*v*x2+u*(1-v)*x3+u*v*x4);
        x1 = in[offset_in1+i*3+1];
        x2 = in[offset_in1+(i+1)*3+1];
        x3 = in[offset_in2+i*3+1];
        x4 = in[offset_in2+(i+1)*3+1];
        out[offset_out+1] = ((1-u)*(1-v)*x1+(1-u)*v*x2+u*(1-v)*x3+u*v*x4);
        x1 = in[offset_in1+i*3+2];
        x2 = in[offset_in1+(i+1)*3+2];
        x3 = in[offset_in2+i*3+2];
        x4 = in[offset_in2+(i+1)*3+2];
        out[offset_out+2] = ((1-u)*(1-v)*x1+(1-u)*v*x2+u*(1-v)*x3+u*v*x4);
    }
}

__global__ void undistored_kernal(unsigned char *in, unsigned char *out, int w, int h, float fx, float fy, float cx, float cy,
    float k1, float k2, float p1, float p2)
{
    int x = blockIdx.x;
    int y = threadIdx.x;
    float u_distorted = 0, v_distorted = 0;
    float x1,y1,x2,y2;
    x1 = (x-cx)/fx;
    y1 = (y-cy)/fy;
    float r2;
    r2 = powf(x1,2)+powf(y1,2);
    x2  = x1*(1+k1*r2+k2*powf(r2,2))+2*p1*x1*y1+p2*(r2+2*x1*x1);
    y2 = y1*(1+k1*r2+k2*powf(r2,2))+p1*(r2+2*y1*y1)+2*p2*x1*y1;
    u_distorted = fx*x2+cx;
    v_distorted = fy*y2+cy;
    int inx = u_distorted, iny = v_distorted;
    int odx = y*w*3+x*3;
    int idx = iny*w*3+inx*3;
    if(inx<0||inx>=w||iny<0||iny>=h)
    {
        out[odx+0] = 0;
        out[odx+1] = 0;
        out[odx+2] = 0;
    }
    else
    {
        out[odx+0] = in[idx+0];
        out[odx+1] = in[idx+1];
        out[odx+2] = in[idx+2];
    }
}

void cudaYUYV2YUV(unsigned char *in, unsigned char *out, int w, int h)
{
    yuyv2yuv_kernal<<<w, h>>>(in,out,w,h);
}

void cudaYUYV2BGR(unsigned char *in, unsigned char *out, int w, int h)
{
    yuyv2bgr_kernal<<<w, h>>>(in,out,w,h);
}

void cudaBayer2BGR(unsigned char *bayer, unsigned char *bgr, int w, int h, 
    float sat, float rgain, float ggain, float bgain)
{
    baygr2bgr_kernal<<<w,h>>>(bayer, bgr, w, h, sat, rgain, ggain, bgain);
}

void cudaBGR2RGBfp(unsigned char *bgr, float *rgbfp, int w, int h)
{
    bgr2rgbfp<<<w,h>>>(bgr, rgbfp, w, h);
}

void cudaResizePacked(float *in, int iw, int ih, float *sized, int ow, int oh)
{
    resize_packed_kernal<<<ow, oh>>>(in, iw, ih, sized, ow, oh);
}

void cudaResizePacked(unsigned char *in, int iw, int ih, unsigned char *sized, int ow, int oh)
{
    resize_packed_kernal<<<ow, oh>>>(in, iw, ih, sized, ow, oh);
}

void cudaUndistored(unsigned char *in, unsigned char *out, int w, int h, float fx, float fy, float cx, float cy,
    float k1, float k2, float p1, float p2)
{
    undistored_kernal<<<w,h>>>(in, out, w, h, fx, fy, cx, cy, k1, k2, p1, p2);
}

